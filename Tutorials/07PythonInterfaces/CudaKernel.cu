
#include <hip/hip_runtime.h>
__global__ void vectorAddition(const float* a, const float* b, float* result, const float scalar)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    result[index] = a[index] + b[index] + scalar;
}
